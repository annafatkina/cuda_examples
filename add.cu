#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>




__global__ void vecAdd(double* res, double* inA, double* inB, size_t n) {
        int x = blockDim.x * blockIdx.x + threadIdx.x;
        if (x >= n) return;
        res[x] = inA[x] + inB[x];
}


void add() {
	size_t N = 1000;
	double *A = new double[N];
	double *B = new double[N];
	double *C = new double[N];
	double *dev_A;
	double *dev_B;
	double *dev_C;
	hipError_t err;
	int alloc_size = N*sizeof(double);

        err = hipMalloc((void**)&dev_A, alloc_size);
        if (err != hipSuccess) {
                printf("ERROR: unable to  allocate!\n");
                std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
        }

        err = hipMalloc((void**)&dev_B, alloc_size);
        if (err != hipSuccess) {
                printf("ERROR: unable to  allocate!\n");
                std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
        }

        err = hipMalloc((void**)&dev_C, alloc_size);
        if (err != hipSuccess) {
                printf("ERROR: unable to  allocate!\n");
                std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
        }

	for(int i = 0; i < N ; i++) {
		A[i] = i;
		B[i]= N*i;
	}
	
	err = hipMemcpy(dev_A, A, alloc_size, hipMemcpyHostToDevice);
        if (err != hipSuccess) {
                printf("ERROR: unable to copy h2d!\n");
                std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
        }
	
	err = hipMemcpy(dev_B, B, alloc_size, hipMemcpyHostToDevice);
        if (err != hipSuccess) {
                printf("ERROR: unable to copy h2d!\n");
                std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
        }
	
	vecAdd<<<N,1>>>(dev_C, dev_A, dev_B, N);
	err = hipMemcpy(C, dev_C, alloc_size, hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
                printf("ERROR: unable to copy h2d!\n");
                std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
        }
	
	for(int i =0;i<N;i++) {
		std::cout << A[i] << " + " << B[i] << " = " << C[i] <<std::endl;
	}
	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_C);
}



int main () {
	add();
	return 0;
}

