#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>




__global__ void vecAdd(double* res, double* inA, double* inB, size_t n) {
        int x = blockDim.x * blockIdx.x + threadIdx.x;
        if (x >= n) return;
        res[x] = inA[x] + inB[x];
}


void add() {
	size_t N = 1000;
	double *A;
	double *B;
	double *C;
	hipError_t err;
	int alloc_size = N*sizeof(double);

        err = hipMallocManaged(&A, alloc_size);
        if (err != hipSuccess) {
                printf("ERROR: unable to  allocate!\n");
                std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
        }

        err = hipMallocManaged(&B, alloc_size);
        if (err != hipSuccess) {
                printf("ERROR: unable to  allocate!\n");
                std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
        }

        err = hipMallocManaged(&C, alloc_size);
        if (err != hipSuccess) {
                printf("ERROR: unable to  allocate!\n");
                std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
        }

	for(int i = 0; i < N ; i++) {
		A[i] = i;
		B[i]= N*i;
	}
	
	vecAdd<<<N,1>>>(C, A, B, N);
	hipDeviceSynchronize();
	
	for(int i =0;i<N;i++) {
		std::cout << A[i] << " + " << B[i] << " = " << C[i] <<std::endl;
	}
}



int main () {
	add();
	return 0;
}

