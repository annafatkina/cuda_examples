#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>




__global__ void vecAdd(double* res, double* inA, double* inB, size_t n) {
        int x = blockDim.x * blockIdx.x + threadIdx.x;
        if (x >= n) return;
        res[x] = inA[x] + inB[x];
}



int main () {
	int* minGridSize = (int*)malloc(sizeof(int));
	int* blockSize = (int*)malloc(sizeof(int));
	hipError_t err = hipOccupancyMaxPotentialBlockSize(minGridSize, blockSize, 
								vecAdd);
	std::cout << "Occupancy minGridSize = " << *minGridSize << ", blockSize = " << *blockSize << std::endl;

	return 0;
}

